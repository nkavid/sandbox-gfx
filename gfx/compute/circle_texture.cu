#include "hip/hip_runtime.h"
#include "circle_texture.cuh"

#include <hip/hip_runtime.h>

__global__ void drawCircleTexture(hipSurfaceObject_t surf)
{
  int x      = (threadIdx.x + blockIdx.x * blockDim.x);
  int y      = (threadIdx.y + blockIdx.y * blockDim.y);
  int radius = 256;

  float4 data = make_float4(0.0f, 0.0f, 0.0f, 0x00);

  int2 pos{x - radius, y - radius};

  if ((pos.x * pos.x + pos.y * pos.y) <= radius * radius)
  {
    data.x = 1.0F;
    data.z = 1.0F;
  }
  else
  {
    data.y = 1.0F;
  }
  surf2Dwrite(data, surf, x * sizeof(float4), y, hipBoundaryModeZero);
}

namespace gfx::compute
{
void callDrawCircleTexture(hipArray_t array, size_t width, size_t height)
{
  hipSurfaceObject_t surfObj;

  HIP_RESOURCE_DESC resDesc;
  memset(&resDesc, 0, sizeof(HIP_RESOURCE_DESC));
  resDesc.resType          = HIP_RESOURCE_TYPE_ARRAY;
  resDesc.res.array.hArray = array;

  cuSurfObjectCreate(&surfObj, &resDesc);

  size_t threadDim = 32;
  dim3 block{static_cast<uint32_t>(width / threadDim),
             static_cast<uint32_t>(height / threadDim),
             1};
  dim3 thread{static_cast<uint32_t>(threadDim), static_cast<uint32_t>(threadDim), 1};

  drawCircleTexture<<<block, thread>>>(surfObj);
}
}
