#include "hip/hip_runtime.h"
#include "circle.cuh"

#include <hip/hip_runtime.h>

__global__ static void drawCircle(uint8_t* array)
{
  int x        = (threadIdx.x + blockIdx.x * blockDim.x);
  int y        = (threadIdx.y + blockIdx.y * blockDim.y);
  size_t pitch = 512 * 4;
  int radius   = 256;

  int2 pos{x - radius, y - radius};

  if ((pos.x * pos.x + pos.y * pos.y) <= radius * radius)
  {
    array[x * 4 + y * pitch + 0] = 0xff;
    array[x * 4 + y * pitch + 2] = 0xff;
  }
  else
  {
    array[x * 4 + y * pitch + 1] = 0xff;
  }
}

namespace gfx::compute
{
void callDrawCircle(hipDeviceptr_t array, size_t width, size_t height)
{
  size_t threadDim = 32;
  dim3 block{static_cast<uint32_t>(width / threadDim),
             static_cast<uint32_t>(height / threadDim),
             1};
  dim3 thread{static_cast<uint32_t>(threadDim), static_cast<uint32_t>(threadDim), 1};
  drawCircle<<<block, thread>>>((uint8_t*)array);
}
} // namespace gfx::compute
