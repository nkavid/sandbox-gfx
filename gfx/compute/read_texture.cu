#include "read_texture.cuh"

#include <stdexcept>

namespace gfx::compute
{
void readFromTexture(hipArray_t array,
                     const Size& arraySize,
                     hipDeviceptr_t devicePtr,
                     const Size& devicePtrSize)
{
  if (arraySize != devicePtrSize)
  {
    throw std::runtime_error("gfx::compute::readFromTexture mismatch size");
  }

  // TODO read from tex2d in device
}
} // namespace gfx::compute
