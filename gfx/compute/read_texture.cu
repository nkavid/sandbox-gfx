#include "hip/hip_runtime.h"
#include "read_texture.cuh"

#include <hip/hip_runtime.h>

#include <stdexcept>

__global__ void readFromTextureKernel(hipTextureObject_t texObj, uint8_t* array)
{
  int x        = (threadIdx.x + blockIdx.x * blockDim.x);
  int y        = (threadIdx.y + blockIdx.y * blockDim.y);
  size_t pitch = 512 * 4;

  auto val = tex2D<int4>(texObj, x, y);

  array[x * 4 + y * pitch + 0] = val.x;
  array[x * 4 + y * pitch + 1] = val.y;
  array[x * 4 + y * pitch + 2] = val.z;
}

namespace gfx::compute
{
void readFromTexture(hipArray_t array,
                     const Size& arraySize,
                     hipDeviceptr_t devicePtr,
                     const Size& devicePtrSize)
{
  if (arraySize != devicePtrSize)
  {
    throw std::runtime_error("gfx::compute::readFromTexture mismatch size");
  }

  hipTextureObject_t texObj;

  HIP_RESOURCE_DESC resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType          = HIP_RESOURCE_TYPE_ARRAY;
  resDesc.res.array.hArray = array;

  HIP_TEXTURE_DESC texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0]      = HIP_TR_ADDRESS_MODE_CLAMP;
  texDesc.addressMode[1]      = HIP_TR_ADDRESS_MODE_CLAMP;
  texDesc.addressMode[2]      = HIP_TR_ADDRESS_MODE_CLAMP;
  texDesc.filterMode          = HIP_TR_FILTER_MODE_POINT;
  texDesc.flags               = HIP_TRSF_READ_AS_INTEGER;
  texDesc.maxAnisotropy       = 1;
  texDesc.mipmapFilterMode    = HIP_TR_FILTER_MODE_LINEAR;
  texDesc.mipmapLevelBias     = 0;
  texDesc.minMipmapLevelClamp = 0;
  texDesc.maxMipmapLevelClamp = 0;

  hipTexObjectCreate(&texObj, &resDesc, &texDesc, nullptr);

  const size_t width{512UL};
  const size_t height{512UL};

  size_t threadDim = 32;
  dim3 block{static_cast<uint32_t>(width / threadDim),
             static_cast<uint32_t>(height / threadDim),
             1};
  dim3 thread{static_cast<uint32_t>(threadDim), static_cast<uint32_t>(threadDim), 1};

  readFromTextureKernel<<<block, thread>>>(texObj, (uint8_t*)devicePtr);
}
} // namespace gfx::compute
